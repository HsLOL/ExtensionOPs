#include "hip/hip_runtime.h"
#include<ATen/ATen.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<vector>

template <typename scalar_t>
__device__ scalar_t sigmoid(scalar_t z){
    return 1.0 / (1.0 + exp(-z));
}

template <typename scalar_t>
__device__ scalar_t d_sigmoid(scalar_t z){
    return (1.0 - z) * z;
}

template <typename scalar_t>
__global__ void sigmoid_cuda_forward_kernel(const scalar_t * __restrict__ input, scalar_t * __restrict__ output){
    const int index = blockIdx.x * blockDim.x + blockIdx.y;
    output[index] = sigmoid(input[index]);
}

template <typename scalar_t>
__global__ void sigmoid_cuda_backward_kernel(const scalar_t* __restrict__ output,
                                             scalar_t* __restrict__ new_grad_output){
    const int index = blockIdx.x * blockDim.x + blockIdx.y;
    new_grad_output[index] = d_sigmoid(output[index]);
}

// only using at::Tensor in .cu file
// not using torch::Tensor
at::Tensor sigmoid_cuda_forward(at::Tensor input){
    auto output = at::zeros_like(input);
    dim3 blocks(input.size(0), input.size(1));
    int threads = 1;

    AT_DISPATCH_FLOATING_TYPES(input.type(), "error in sigmoid_cuda_forward", ([&]
        {sigmoid_cuda_forward_kernel<scalar_t> <<<blocks, threads>>> (input.data<scalar_t>(), output.data<scalar_t>());
        }));

    return output;
}

at::Tensor sigmoid_cuda_backward(at::Tensor output){
    auto new_grad_output = at::zeros_like(output);
    dim3 blocks(output.size(0), output.size(1));
    int threads = 1;

    AT_DISPATCH_FLOATING_TYPES(output.type(), "error in sigmoid_cuda_backward", ([&]{
    sigmoid_cuda_backward_kernel<scalar_t> <<<blocks, threads>>> (output.data<scalar_t>(),
                                                                  new_grad_output.data<scalar_t>());}));

    return new_grad_output;
}
